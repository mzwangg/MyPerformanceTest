#include "hip/hip_runtime.h"
#pragma once
#include <string.h>
#include <opencv2/opencv.hpp>
#include "config.h"
#include "hip/hip_runtime.h"
#include ""

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

//random_device rd;
//default_random_engine eng(rd());
//uniform_real_distribution<double> distr(0, 100);

//template<typename T>
//void initialData(T* ip, int size)
//{
//    for (int i = 0; i < size; i++)
//    {
//        ip[i] = (T)distr(eng);
//    }
//}

template<typename T>
void printMatrix(T* C, const int nx, const int ny)
{
    T* ic = C;
    printf("Matrix<%d,%d>:", ny, nx);
    for (int i = 0; i < ny; i++)
    {
        for (int j = 0; j < nx; j++)
        {
            printf("%6f ", ic[j]);
        }
        ic += nx;
        printf("\n");
    }
}

void initDevice(int devNum)
{
    int dev = devNum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    if (GPU_CHECK)printf("Using device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
}

template<typename T>
void checkResult(T* hostRef, T* gpuRef, const int N)
{
    double eps = 1e-8;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > eps)
        {
            printf("Results don\'t match!\n");
            printf("%f(hostRef[%d] )!= %f(gpuRef[%d])\n", hostRef[i], i, gpuRef[i], i);
            return;
        }
    }
    printf("Check result success!\n");
}

template<typename T>
void matrixMulCPU(T* A, T* B, T* C, int n)
{
    memset(C, 0, n * n * sizeof(T));
    for (int i = 0; i < n; i++)
    {
        int ik = i * n;
        for (int k = 0; k < n; k++)
        {
            int ij = i * n;
            int kj = k * n;
            for (int j = 0; j < n; j++)
            {
                C[ij] += A[ik] * B[kj];
                ij++;
                kj++;
            }
            ik++;
        }
    }
}

template<typename T>
__global__ void matrixMulGPU(T* A, T* B, T* C, int n)
{
    int nRow = blockIdx.x;
    int nCol = threadIdx.x;
    T fCVal = 0;

    for (int i = 0; i < n; i++)
    {
        fCVal += A[nRow * n + i] * B[i * n + nCol];
    }

    C[nRow * n + nCol] = fCVal;
}

template<typename T>
double gpuCalculateTest(int n, T* A_host, T* B_host, T* C_from_gpu, T* C_host = NULL)
{
    initDevice(0);
    int nxy = n * n;
    int nBytes = nxy * sizeof(T);

    //hipMalloc
    T* A_dev = NULL;
    T* B_dev = NULL;
    T* C_dev = NULL;
    CHECK(hipMalloc((void**)&A_dev, nBytes));
    CHECK(hipMalloc((void**)&B_dev, nBytes));
    CHECK(hipMalloc((void**)&C_dev, nBytes));
    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    // 1d block and 1d grid
    dim3 block(n);
    dim3 grid((nxy - 1) / block.x + 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrixMulGPU<T> << <grid, block >> > (A_dev, B_dev, C_dev, n);
    CHECK(hipDeviceSynchronize());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    if (GPU_CHECK)
    {
        printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
            grid.x, grid.y, block.x, block.y, time);
        matrixMulCPU<T>(A_host, B_host, C_host, n);
        CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
        checkResult(C_host, C_from_gpu, nxy);
    }

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    hipDeviceReset();
    return time / 1000.0;
}

template<typename T>
double gpuCalTypeTest()
{
    int nBytes = GPU_N * GPU_N * sizeof(T);
    T* A = (T*)malloc(nBytes);
    T* B = (T*)malloc(nBytes);
    T* C_from_gpu = (T*)malloc(nBytes);

    //initialData<T>(A, GPU_N * GPU_N);
    //initialData<T>(B, GPU_N * GPU_N);

    double time = 0;
    if (GPU_CHECK)
    {
        T* C = (T*)malloc(nBytes);
        time += gpuCalculateTest<T>(GPU_N, A, B, C_from_gpu, C);
        free(C);
    }
    else
    {
        for (int n = 1 << 6; n <= GPU_N; n <<= 1)
        {
            time += gpuCalculateTest<T>(n, A, B, C_from_gpu) * pow(GPU_N / n, 3);
        }
    }

    free(A);
    free(B);
    free(C_from_gpu);
    return time;
}

__global__ void gaussianBlur(unsigned char* src, unsigned char* dst, int width, int height, float sigma)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height)
    {
        float sum = 0.0f;
        float pixel = 0.0f;
        float weight = 0.0f;

        for (int j = -2; j <= 2; j++)
        {
            for (int i = -2; i <= 2; i++)
            {
                int xIndex = x + i;
                int yIndex = y + j;

                if (xIndex >= 0 && yIndex >= 0 && xIndex < width && yIndex < height)
                {
                    pixel = src[yIndex * width + xIndex];
                    weight = exp(-(i * i + j * j) / (2 * sigma * sigma)) / (3.1415926f * 2 * sigma * sigma);
                    sum += weight * pixel;
                }
            }
        }
        dst[y * width + x] = (unsigned char)sum;
    }
}

double gpuGraphtest()
{
    float time = 0;
    for (int step = 0; step < 10; step++)
    {
        for (int i = 1; i <= 5; i++)
        {
            string inputPath = "InputPic/" + to_string(i) + ".png";
            cv::Mat image = cv::imread(inputPath, cv::IMREAD_GRAYSCALE);

            int width = image.cols;
            int height = image.rows;
            int size = width * height * sizeof(unsigned char);

            unsigned char* h_inputImage = (unsigned char*)image.data;
            unsigned char* h_outputImage = (unsigned char*)malloc(size);
            unsigned char* d_inputImage;
            unsigned char* d_outputImage;

            hipMalloc((void**)&d_inputImage, size);
            hipMalloc((void**)&d_outputImage, size);
            hipMemcpy(d_inputImage, h_inputImage, size, hipMemcpyHostToDevice);

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            gaussianBlur << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, width, height, SIGMA);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float localTime = 0;
            hipEventElapsedTime(&localTime, start, stop);

            hipFree(d_inputImage);
            hipFree(d_outputImage);
            free(h_outputImage);
            time += localTime;
        }
    }
    return (double)time / 1000.0;
}

extern "C" double gpuTest()
{
    //�������岿��
    double intTime = gpuCalTypeTest<int>();
    double doubleTime = gpuCalTypeTest<double>();
    double graphTime = gpuGraphtest();

    double intScore = GPU_INT_BASE_TIME / intTime;
    double doubleScore = GPU_DOUBLE_BASE_TIME / doubleTime;
    double graphScore = GPU_GRAPH_BASE_TIME / graphTime;

    double gpuScore = pow(intScore * doubleScore * graphScore, 1.0 / 3);
    return gpuScore;
}